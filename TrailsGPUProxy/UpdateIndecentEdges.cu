#include "hip/hip_runtime.h"
#include "UpdateIndecentEdges.h"

#ifndef __HIPCC__ // visual studio doesn't show functions declared in device_atomic_function.h without this
#define UNDEF__CUDACC__
#define __HIPCC__
#endif
#include <math_functions.h>
#include <hip/device_functions.h>
#ifdef UNDEF__CUDACC__
#undef UNDEF__CUDACC__
#undef __HIPCC__
#endif

#define BLOCK_SIZE_X UPDATE_INDECENT_EDGES_BLOCK_SIZE_X
#define BLOCK_SIZE_Y UPDATE_INDECENT_EDGES_BLOCK_SIZE_Y


namespace TrailEvolutionModelling {
	namespace GPUProxy {

		inline __device__ void Update(float original, float current, float& target) {
			target = min(original, current);
		}

		__global__ void UpdateIndecentEdgesKernel(EdgesWeightsDevice original,
			EdgesWeightsDevice current, EdgesWeightsDevice target, int graphW, int graphH) 
		{
			int i = blockIdx.x * blockDim.x + threadIdx.x;
			int j = blockIdx.y * blockDim.y + threadIdx.y;
			if(i > graphW || j > graphH)
				return;

			int idx = i + j * (graphW + 1);
			Update(original.horizontal[idx], current.horizontal[idx], target.horizontal[idx]);
			Update(original.vertical[idx], current.vertical[idx], target.vertical[idx]);
			Update(original.leftDiagonal[idx], current.leftDiagonal[idx], target.leftDiagonal[idx]);
			Update(original.rightDiagonal[idx], current.rightDiagonal[idx], target.rightDiagonal[idx]);
		}

		hipError_t UpdateIndecentEdges(EdgesWeightsDevice* edgesIndecentOriginal, 
			EdgesWeightsDevice* currentEdgesWeights, EdgesWeightsDevice* target, int graphW, int graphH) {
			dim3 threadsDim(BLOCK_SIZE_X, BLOCK_SIZE_Y);
			dim3 blocksDim(GetUpdateIndecentEdgesBlocksX(graphW),
						   GetUpdateIndecentEdgesBlocksY(graphH));

			UpdateIndecentEdgesKernel<<<blocksDim, threadsDim>>>(*edgesIndecentOriginal, 
				*currentEdgesWeights, *target, graphW, graphH);

			return hipGetLastError();
		}

	}
}